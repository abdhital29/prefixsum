#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    }

// Step 1: Mark non-zero flags
__global__ void mark_flags(const int* input, int* flags, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        flags[i] = input[i] != 0 ? 1 : 0;
}

// Step 2: Naive exclusive scan on flags
__global__ void scan_kernel(int* output, const int* input, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n) return;

    int sum = 0;
    for (int j = 0; j < i; ++j)
        sum += input[j];
    output[i] = sum;
}

// Step 3: Scatter elements
__global__ void scatter(const int* input, const int* flags, const int* scanned_flags, int* output, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && flags[i] == 1)
        output[scanned_flags[i]] = input[i];
}

// Host function
void stream_compaction(const std::vector<int>& input, std::vector<int>& compacted, int& valid_count) {
    int n = input.size();
    int* d_input = nullptr;
    int* d_flags = nullptr;
    int* d_scanned = nullptr;
    int* d_output = nullptr;

    CHECK_CUDA(hipMalloc(&d_input, n * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_flags, n * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_scanned, n * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_output, n * sizeof(int)));

    CHECK_CUDA(hipMemcpy(d_input, input.data(), n * sizeof(int), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    mark_flags<<<blocks, threads>>>(d_input, d_flags, n);
    scan_kernel<<<blocks, threads>>>(d_scanned, d_flags, n);
    scatter<<<blocks, threads>>>(d_input, d_flags, d_scanned, d_output, n);

    // Get number of valid (non-zero) elements
    CHECK_CUDA(hipMemcpy(&valid_count, &d_scanned[n - 1], sizeof(int), hipMemcpyDeviceToHost));
    int last_flag;
    CHECK_CUDA(hipMemcpy(&last_flag, &d_flags[n - 1], sizeof(int), hipMemcpyDeviceToHost));
    valid_count += last_flag;

    compacted.resize(valid_count);
    CHECK_CUDA(hipMemcpy(compacted.data(), d_output, valid_count * sizeof(int), hipMemcpyDeviceToHost));

    hipFree(d_input);
    hipFree(d_flags);
    hipFree(d_scanned);
    hipFree(d_output);
}
